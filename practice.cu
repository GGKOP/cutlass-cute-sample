#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "util.h"

using namespace cute;

template <typename T>
void gen_rand_data(T *data, int n)
{
  for (int i = 0; i < n; ++i)
  {
    float v = (rand() % 200 - 100) * 0.01;
    data[i] = v;
  }
}


template <typename T,int kTileM,int kTileN,int ktileK,typename TiledMMA>
__global__ void gemm_simple_cute(T *Cptr, const T *Aptr, const T *Bptr, int m, int n, int k) {

    //进行定义一个大的tensor 这个tensor的主要目的是为了后续的分块放入全局内存。
    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor C = make_tensor(make_gmem_ptr(Cptr), make_shape(m, n), make_stride(n, Int<1>{})); 

    int ix = blockIdx.x;
    int iy = blockIdx.y;
    Tensor gA = local_tile(A, make_tile(Int<kTileM>{}, Int<kTileK>{}), make_coord(iy, _));
    Tensor gB = local_tile(B, make_tile(Int<kTileN>{}, Int<kTileK>{}), make_coord(ix, _));
    Tensor gC = local_tile(C, make_tile(Int<kTileM>{}, Int<kTileN>{}), make_coord(iy, ix));
    //  gA(kTileM, kTileK, num_tile_k)
    //  gB(kTileN, kTileK, num_tile_k)
    //  gC(kTileM, kTileN) 

    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
    auto tAgA = thr_mma.partition_A(gA);  // (MMA, MMA_M, MMA_K, num_tile_k)
    auto tBgB = thr_mma.partition_B(gB);  // (MMA, MMA_N, MMA_K, num_tile_k)
    auto tCgC = thr_mma.partition_C(gC);  // (MMA, MMA_M, MMA_N)
    

    //移动到寄存器
    auto tArA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
    auto tBrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
    auto tCrC = thr_mma.partition_fragment_C(gC(_, _));     // (MMA, MMA_M, MMA_N)
    clear(tcrc);

    int num_tile_k=size<2>(gA);
    for(int itile = 0; itile < num_tile_k; ++itle) {
    cute::copy(tAgA(_, _, _, itile), tArA);
    cute::copy(tBgB(_, _, _, itile), tBrB);
    cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
  }


    cute::copy(tCrC, tCgC); 


}


template <typename T,int kTileM,int kTileN,int ktileK >
__global__ void gemm_simple(T *Cptr, const T *Aptr, const T *Bptr, int m, int n, int k) {

  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int by = blockIdx.y;


  int A_offset = (blockIdx.x * k * kTileM);
  int B_offset = (blockIdx.y * k * kTileN);

  extren__shared__float sram[];

  float* tile_A=sram;
  float* tile_B=&sram[kTileM * k];
  float* tile_C=&sram[kTileM * k + kTileN * k];


  for(int x=0;x<k;x++){
    tile_A[(tx*k)+x] = A[A_offset + (tx * k) + x];
    tile_B[(tx*k)+x] = B[B_offset + (tx * k) + x];
  }

  __syncthreads();

  #pragma unroll
  for(int i=0;i<kTileN;i++){
    float sum =0;
     for(int j=0;j<k;j++){
      sum += tile_A[(tx*k)+j] * tile_B[(i * k)+j];
     }
     tile_C[(tx * kTileN) + i]=sum;
  }

  __syncthreads();

}


template<typename T,int kTileM,int kTileN,int ktileK>
__global__ void gemm_simple(T *Cptr, const T *Aptr, const T *Bptr, int m, int n, int k) {




}













int main(){
  srand(1000);

  using T = cute::half_t;
  hipEvent_t start, end;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&end);
  T *Cptr;
  T *Aptr;
  T *Bptr;

  int m = 1024*64;
  int n = 128;
  int k = 1024;

  hipMalloc(&Cptr, sizeof(T) * m * n);
  hipMalloc(&Aptr, sizeof(T) * m * k);
  hipMalloc(&Bptr, sizeof(T) * k * n);

  T *Aptr_host;
  T *Bptr_host;
  Aptr_host = (T *)malloc(sizeof(T) * m * k);
  Bptr_host = (T *)malloc(sizeof(T) * n * k);
  gen_rand_data(Aptr_host, m * k);
  gen_rand_data(Bptr_host, n * k);

  hipMemcpy(Aptr, Aptr_host, sizeof(T) * m * k, hipMemcpyHostToDevice);
  hipMemcpy(Bptr, Bptr_host, sizeof(T) * n * k, hipMemcpyHostToDevice);

    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits=MMA_Traits<mma_op>;
    using mma_atom =MMA_Atom<mma_traits>;

auto MMA = decltype(make_tiled_mma(mma_atom{}, 
                      make_layout(Shape<_2, _2, _1>{}), 
                      make_layout(Shape<_1, _2, _1>{})));


  PRINT("mma",size(MMA{}));

  constexpr int kTileM = 128;
  constexpr int kTileN = 128;
  constexpr int kTileK = 32;

  // each thread block handle with (kTileM, kTileN) output
  dim3 grid(n / kTileN, m / kTileM);
  dim3 block(size(MMA{}));

  int count = 100;
  hipEventRecord(start);
  for (int i = 0; i < count; ++i)
  {
    gemm_simple_cute<T, kTileM, kTileN, kTileK, MMA><<<grid, block>>>(Cptr, Aptr, Bptr, m, n, k);
  }
  auto err = hipGetLastError();
  printf("err = %d, str = %s\n", err, hipGetErrorString(err));
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsedTime, start, end);
  std::cout << "gemm-simple-cute took " << elapsedTime / count << "ms." << std::endl;


  dim3 grid(n / kTileN, m / kTileM);
  dim3 block(kTileM);
  //const int sram
  int count = 100;
  hipEventRecord(start);
  for (int i = 0; i < count; ++i)
  {
    gemm_simple<T, kTileM, kTileN, kTileK><<<grid, block>>>(Cptr, Aptr, Bptr, m, n, k);
  }
  auto err = hipGetLastError();
  printf("err = %d, str = %s\n", err, hipGetErrorString(err));
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsedTime, start, end);
  std::cout << "gemm-simple took " << elapsedTime / count << "ms." << std::endl;

}