#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "util.h"

using namespace cute;

#define PRINT(name, content) \
    print(name);             \
    print(" : ");            \
    print(content);          \
    print("\n");

template <typename T, typename MMA, int M, int N, int K>
__global__ void mma_simple(T *Cptr, const T *Aptr, const T *Bptr)
{
    MMA tiled_mma;

    // ThrMMA
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);

    // tensor layout should be in static
    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(Int<M>{}, Int<K>{}), make_stride(Int<K>{}, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(Int<N>{}, Int<K>{}), make_stride(Int<K>{}, Int<1>{}));
    Tensor C = make_tensor(make_gmem_ptr(Cptr), make_shape(Int<M>{}, Int<N>{}), make_stride(Int<N>{}, Int<1>{}));

    auto tAgA = thr_mma.partition_A(A);
    auto tBgB = thr_mma.partition_B(B);
    auto tCgC = thr_mma.partition_C(C);

    auto tArA = thr_mma.partition_fragment_A(A);
    auto tBrB = thr_mma.partition_fragment_B(B);
    auto tCrC = thr_mma.partition_fragment_C(C);

    if (threadIdx.x == 0)
    {
        // MMA 由MMA指令决定，不受MMAThrLayout和MMAValLayout影响
        // A,B,C 对应为： 16*16/32=8=(2,2,2), 16*8/32=4=(2,2), 16*8/32=4=(2,2)

        // MMA_M, MMA_K, MMA_N 由MMA指令、MMAThrLayout和源Tensor shape决定，不受MMAValLayout影响
        // MMA_M = M / (mma_op_m * thr_layout_m)
        // MMA_N = N / (mma_op_n * thr_layout_n)
        // MMA_K = K / (mma_op_k * thr_layout_k)

        // (MMA, MMA_M, MMA_K)
        PRINT("tArA.shape", tArA.shape());
        // (MMA, MMA_N, MMA_K)
        PRINT("tBrB.shape", tBrB.shape());
        // (MMA, MMA_M, MMA_N)
        PRINT("tCrC.shape", tCrC.shape());
    }

    cute::copy(tAgA, tArA);
    cute::copy(tBgB, tBrB);
    clear(tCrC);

    // cute::gemm, warp level
    // 语义：处理tCrC, tArA, tBrB 对应的partition前tensor A,B,C的 C= A*B+C, 内部会拆解成大量的mma_atom指令
    cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);

    // thread level
    cute::copy(tCrC, tCgC);
}

/*
Profile
thr_layout,val_layout
thr_layout比较影响速度


M=128, N=128, K=32
Shape<_1, _1, _1>  Shape<_1, _1, _1>
0.040183ms

Shape<_2, _2, _2>  Shape<_1, _1, _1>
0.037417ms

Shape<_4, _4, _2>  Shape<_1, _1, _1>
0.0368947ms

Shape<_4, _4, _2>  Shape<_2, _1, _1>
0.0366608ms

Shape<_4, _4, _2>  Shape<_2, _4, _1>
0.0392909ms

------------------------------------------

M=256, N=256, K=128
Shape<_1, _1, _1>  Shape<_1, _1, _1>
0.751002ms

Shape<_2, _2, _2>  Shape<_1, _1, _1>
0.251904ms

Shape<_4, _4, _2>  Shape<_1, _1, _1>
0.214118ms

Shape<_2, _4, _4>  Shape<_1, _1, _1>
0.206029ms

Shape<_2, _4, _4>  Shape<_2, _2, _2>
0.205395ms

Shape<_2, _4, _4>  Shape<_4, _4, _4>
0.239418ms

*/

int main()
{

    hipEvent_t start, end;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&end);

    using T = cute::half_t;
    // MMAOperation, M=16, N=8, K=16, type=half
    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    // MMA_Traits
    using mma_traits = MMA_Traits<mma_op>;
    // MMA_ATOM
    using mma_atom = MMA_Atom<mma_traits>;
    // TiledMMA
    // using MMA = decltype(make_tiled_mma(mma_atom{},
    //                                     make_layout(Shape<_1, _1, _1>{}),   // thr_layout
    //                                     make_layout(Shape<_1, _1, _1>{}))); // val_layout

    using MMA = decltype(make_tiled_mma(mma_atom{},
                                        make_layout(Shape<_2, _4, _4>{}),   // thr_layout
                                        make_layout(Shape<_4, _4, _4>{}))); // val_layout
    // constexpr int M = 128;
    // constexpr int N = 128;
    // constexpr int K = 32;

    constexpr int M = 256;
    constexpr int N = 256;
    constexpr int K = 128;

    T *Cptr;
    T *Aptr;
    T *Bptr;
    hipMalloc(&Cptr, sizeof(T) * M * N);
    hipMalloc(&Aptr, sizeof(T) * M * K);
    hipMalloc(&Bptr, sizeof(T) * K * N);

    dim3 block(size(MMA{}));
    print(size(MMA{}));
    print("\n");
    hipEventRecord(start);
    int count = 10;
    for (int i = 0; i < count; ++i)
    {
        mma_simple<T, MMA, M, N, K><<<1, block>>>(Cptr, Aptr, Bptr);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsedTime, start, end);
    std::cout << "mma_simple took " << elapsedTime / count << "ms." << std::endl;
}